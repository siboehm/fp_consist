#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <sys/time.h>
// #include <helper_functions.h>

#define VECTORDIM 3
typedef float mt;

double cpuSecond() {
  struct timeval tp;
  gettimeofday(&tp, NULL);
  return ((double)tp.tv_sec + (double)tp.tv_usec * 1e-6);
}

void DistanceCPU(mt *array1, mt *array2, int narray1, int narray2, mt *output) {
  mt temp;
  for (int i = 0; i < narray1; i++) {
    for (int j = 0; j < narray2; j++) {
      temp = 0;
      for (int l = 0; l < VECTORDIM; l++) {
#ifndef USE_POW
        temp += (array1[i + l * narray1] - array2[j + l * narray2]) *
                (array1[i + l * narray1] - array2[j + l * narray2]);
#else
        temp += powf(array1[i + l * narray1] - array2[j + l * narray2], 2);
#endif
      }
      output[i * narray2 + j] = temp;
    }
  }
}
__global__ void DistGPU(mt *array1, mt *array2, int narray1, int narray2,
                        mt *output) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  mt temp;

  if (i < narray1) {
    for (int j = 0; j < narray2; j++) {
      temp = 0;
#ifndef USE_POW
      temp += (array1[i] - array2[j]) * (array1[i] - array2[j]);
      temp += (array1[i + narray1] - array2[j + narray2]) *
              (array1[i + narray1] - array2[j + narray2]);
      temp += (array1[i + 2 * narray1] - array2[j + 2 * narray2]) *
              (array1[i + 2 * narray1] - array2[j + 2 * narray2]);
#else
      temp += powf(array1[i] - array2[j], 2);
      temp += powf(array1[i + narray1] - array2[j + narray2], 2);
      temp += powf(array1[i + 2 * narray1] - array2[j + 2 * narray2], 2);
#endif
      output[i * narray2 + j] = temp;
    }
  }
}

int main() {
  int narray1 = 7000;
  int narray2 = 60000;

  mt *array1 = new mt[narray1 * VECTORDIM];
  mt *array2 = new mt[narray2 * VECTORDIM];
  mt *outputGPU = new mt[narray1 * narray2];
  mt *outputCPU = new mt[narray1 * narray2];
  mt *outputCPUTest = new mt[narray1 * narray2];

  mt *d_array1;
  mt *d_array2;
  mt *d_output;

  for (int i = 0; i < narray1 * VECTORDIM; i++) {
    array1[i] = static_cast<mt>(rand() / (static_cast<mt>(RAND_MAX / 10)));
    // std::cout << "Element " << i << " " << array1[i] << std::endl;
  }

  for (int i = 0; i < narray2 * VECTORDIM; i++) {
    array2[i] = static_cast<mt>(rand() / (static_cast<mt>(RAND_MAX / 10)));
  }

  hipError_t err;

  err = hipMalloc((void **)&d_array1, narray1 * VECTORDIM * sizeof(mt));
  err = hipMalloc((void **)&d_array2, narray2 * VECTORDIM * sizeof(mt));
  err = hipMalloc((void **)&d_output, narray1 * narray2 * sizeof(mt));

  err = hipMemcpy(d_array1, array1, narray1 * VECTORDIM * sizeof(mt),
                   hipMemcpyHostToDevice);
  err = hipMemcpy(d_array2, array2, narray2 * VECTORDIM * sizeof(mt),
                   hipMemcpyHostToDevice);

  int threadsPerBlock = 512;
  int blocksPerGrid = (narray1 + threadsPerBlock - 1) / threadsPerBlock;
  printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
         threadsPerBlock);
  double iStart = cpuSecond();
  DistGPU<<<blocksPerGrid, threadsPerBlock>>>(d_array1, d_array2, narray1,
                                              narray2, d_output);
  double iElaps = cpuSecond() - iStart;

  err = hipMemcpy(outputGPU, d_output, narray1 * narray2 * sizeof(mt),
                   hipMemcpyDeviceToHost);

  printf("Total computation time is %lf \n", iElaps);

  DistanceCPU(array1, array2, narray1, narray2, outputCPU);

  mt error = 0;
  bool bitequal = true;
  for (long i = 0; i < narray1 * narray2; i++) {
    error += abs(outputCPU[i] - outputGPU[i]);
    if (outputCPU[i] != outputGPU[i]) {
      bitequal = false;
    }
  }
  error /= (narray2 * narray1);

  for (int i = 0; i < 20; i++) {
    printf("CPU result %f \n", outputCPU[i]);
    printf("GPU result %f \n", outputGPU[i]);
  }

  printf("Error is %f\n", error);
  printf("Are the results bitequal? %s\n", bitequal ? "Yes" : "No");
  delete[] array1;
  delete[] array2;
  delete[] outputCPU;
  delete[] outputGPU;
  return 0;
}
